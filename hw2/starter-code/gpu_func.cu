#include "hip/hip_runtime.h"
#define ARMA_ALLOW_FAKE_GCC
#include <algorithm>
#include <armadillo>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include <memory>

#include "gpu_func.h"

// +-*=+-*=+-*=+-*=+-*=+-*=+-*=+-*=+*-=+-*=+*-=+-*=+-*=+-*=+-*=+-*= //
//                          DeviceAllocator 						//
// +-*=+-*=+-*=+-*=+-*=+-*=+-*=+-*=+*-=+-*=+*-=+-*=+-*=+-*=+-*=+-*= //

DeviceAllocator::DeviceAllocator(nn_real *cpu_data, int n)
{
  // TODO: implement this constructor
  this->nbytes = n * sizeof(nn_real);
  // Allocate memory on the GPU and copy the CPU data to the GPU.
  checkCudaErrors(hipMalloc(&this->data, this->nbytes));
  checkCudaErrors(hipMemcpy(this->data, cpu_data, this->nbytes, hipMemcpyHostToDevice));
}

DeviceAllocator::DeviceAllocator(int n)
{
  // TODO: implement this constructor
  // Only allocate memory on the GPU.
  checkCudaErrors(hipMalloc(&this->data, n * sizeof(nn_real)));
  this->nbytes = n * sizeof(nn_real);
}

DeviceAllocator::~DeviceAllocator()
{
  // TODO: implement this destructor
  // Deallocate the memory on the GPU.
  checkCudaErrors(hipFree(this->data));
}

void DeviceAllocator::to_cpu(nn_real *cpu_data)
{
  // TODO: implement this function
  // Copy the GPU data to the CPU pointer.
  checkCudaErrors(hipMemcpy(cpu_data, this->data, this->nbytes, hipMemcpyDeviceToHost));
}

// +-*=+-*=+-*=+-*=+-*=+-*=+-*=+-*=+*-=+-*=+*-=+-*=+-*=+-*=+-*=+-*= //
//                          DeviceMatrix 							//
// +-*=+-*=+-*=+-*=+-*=+-*=+-*=+-*=+*-=+-*=+*-=+-*=+-*=+-*=+-*=+-*= //

DeviceMatrix::DeviceMatrix(int n_rows, int n_cols)
{
  // TODO: implement this constructor
  this->n_rows = n_rows;
  this->n_cols = n_cols;
  this->allocator = std::make_unique<DeviceAllocator>(n_rows * n_cols);
  this->data = this->allocator->data;
}

DeviceMatrix::DeviceMatrix(arma::Mat<nn_real> &cpu_mat)
{
  // TODO: implement this constructor
  this->n_rows = cpu_mat.n_rows;
  this->n_cols = cpu_mat.n_cols;
  this->allocator = std::make_unique<DeviceAllocator>(cpu_mat.memptr(), cpu_mat.n_elem);
  this->data = this->allocator->data;
}

void DeviceMatrix::to_cpu(arma::Mat<nn_real> &cpu_mat)
{
  this->allocator->to_cpu(cpu_mat.memptr());
}

__device__ nn_real &DeviceMatrix::operator()(int row, int col)
{
  // Note that arma matrices are column-major
  return data[col * this->n_rows + row];
}

int DeviceMatrix::total_bytes()
{
  return allocator->nbytes;
}

// +-*=+-*=+-*=+-*=+-*=+-*=+-*=+-*=+*-=+-*=+*-=+-*=+-*=+-*=+-*=+-*= //
//                           CUDA kernels                           //
// +-*=+-*=+-*=+-*=+-*=+-*=+-*=+-*=+*-=+-*=+*-=+-*=+-*=+-*=+-*=+-*= //

/**
 * A CUDA kernel function that applies the sigmoid function element-wise to a
 * matrix on the GPU.
 *
 * @param src The input matrix.
 * @param dst The output matrix.
 */
__global__ void MatSigmoid(DeviceMatrix src, DeviceMatrix dst)
{
  // TODO: implement this kernel function
  // Hint: Use Exp() from common.h
  // Sigmod = 1 / (1 + exp(-x))
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
          i < src.n_rows;
          i += blockDim.x * gridDim.x) {
            for (int j = blockIdx.y * blockDim.y + threadIdx.y;
                    j < src.n_cols;
                    j += blockDim.y * gridDim.y) {
                      dst(i, j) = 1 / (1 + Exp(-src(i, j)));
                    }
          }
  // if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
  //   printf("dst(0, 0) = %f, dst(0, 1) = %f\n", dst(0, 0), dst(0, 1));
  // }
}

/**
 * A CUDA kernel function that repeats each column of the source matrix `repeat`
 * times and stores the result in the destination matrix.
 *
 * @param src The source matrix to repeat.
 * @param dst The destination matrix to store the repeated columns.
 * @param repeat The number of times to repeat each column.
 */
__global__ void MatRepeatColVec(DeviceMatrix src, DeviceMatrix dst,
                                int repeat)
{
  // TODO: implement this kernel function
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
          i < src.n_rows;
          i += blockDim.x * gridDim.x) {
            for (int j = blockIdx.y * blockDim.y + threadIdx.y;
                    j < repeat;
                    j += blockDim.y * gridDim.y) {
                      dst(i, j) = src(i, 0);
                    }
          }
}

/**
 * A CUDA kernel function that computes the sum of a matrix along a specified
 * axis on the GPU.
 *
 * @param src The input matrix.
 * @param dst The output matrix.
 * @param alpha The scaling factor for the sum.
 * @param axis The axis along which to compute the sum (0 for rows, 1 for
 * columns).
 */
__global__ void MatSum(DeviceMatrix src, DeviceMatrix dst, nn_real alpha,
                       int axis)
{
  // TODO: implement this kernel function
  if (axis == 0) {
    for (int j = blockIdx.x * blockDim.x + threadIdx.x;
            j < src.n_cols;
            j += blockDim.x * gridDim.x) {
              nn_real sum = 0;
              for (int i = 0; i < src.n_rows; i++) {
                sum += src(i, j);
              }
              dst(0, j) = alpha * sum;
            }
  }
  else {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
            i < src.n_rows; 
            i += blockDim.x * gridDim.x) {
              nn_real sum = 0;
              for (int j = 0; j < src.n_cols; j++) {
                sum += src(i, j);
              }
              dst(i, 0) = alpha * sum;
            }
  }
}

/**
 * A CUDA kernel function that applies the softmax function along a specified
 * axis to a matrix on the GPU.
 *
 * @param src The input matrix.
 * @param dst The output matrix.
 * @param axis The axis along which to apply the softmax function (0 for rows, 1
 * for columns).
 */
__global__ void MatSoftmax(DeviceMatrix src, DeviceMatrix dst, int axis)
{
  /**
   * TODO: implement this kernel function
   * Hint: Use Exp() from common.h
   * A possible implementation is to have one thread per row (or  column,
   * depending on axis), compute the sum of exponentials of all elements in
   * the row by iterating through elements in the row, and then replace
   * dst(row, col) with the exponential of src(row, col) divided by the sum.
   */
   if (axis == 0)
   {
      for (int j = blockIdx.x * blockDim.x + threadIdx.x;
              j < src.n_cols;
              j += blockDim.x * gridDim.x) {
                nn_real sum = 0;
                for (int i = 0; i < src.n_rows; i++) {
                  sum += Exp(src(i, j));
                }
                for (int i = 0; i < src.n_rows; i++) {
                  dst(i, j) = Exp(src(i, j)) / sum;
                }
              }
   }
   else
   {
      for (int i = blockIdx.x * blockDim.x + threadIdx.x;
              i < src.n_rows;
              i += blockDim.x * gridDim.x) {
                nn_real sum = 0;
                for (int j = 0; j < src.n_cols; j++) {
                  sum += Exp(src(i, j));
                }
                for (int j = 0; j < src.n_cols; j++) {
                  dst(i, j) = Exp(src(i, j)) / sum;
                }
              }
   }
}

/**
 * A CUDA kernel function that computes the cross-entropy loss between predicted
 * and true labels on the GPU.
 *
 * @param y_pred The predicted label matrix.
 * @param y The true label matrix.
 * @param loss The output loss matrix.
 */
__global__ void MatCrossEntropyLoss(DeviceMatrix y_pred, DeviceMatrix y,
                                    DeviceMatrix loss)
{
  /**
   * TODO: implement this kernel function
   * Hint: This kernel computes loss = -y * log(y_pred) where * denotes
   * element-wise multiplication and log is applied element-wise. Use
   * Log() from common.h
   */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
          i < y_pred.n_rows;
          i += blockDim.x * gridDim.x) {
            for (int j = blockIdx.y * blockDim.y + threadIdx.y;
                    j < y_pred.n_cols;
                    j += blockDim.y * gridDim.y) {
                      loss(i, j) = -y(i, j) * Log(y_pred(i, j));
                    }
          }
}

/**
 * A CUDA kernel function that performs element-wise arithmetic operations on
 * two matrices on the GPU. A = alpha * (A + beta * B)
 *
 * @param A The first input matrix.
 * @param B The second input matrix.
 * @param alpha The scaling factor for the first input matrix.
 * @param beta The scaling factor for the second input matrix.
 */
__global__ void MatElemArith(DeviceMatrix A, DeviceMatrix B, nn_real alpha,
                             nn_real beta)
{
  // TODO: implement this kernel function
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
          i < A.n_rows;
          i += blockDim.x * gridDim.x) {
            for (int j = blockIdx.y * blockDim.y + threadIdx.y;
                    j < A.n_cols;
                    j += blockDim.y * gridDim.y) {
                      A(i, j) = alpha * (A(i, j) + beta * B(i, j));
                    }
          }
}

/**
 * A CUDA kernel function that computes the element-wise square of a matrix on
 * the GPU.
 *
 * @param src The input matrix.
 * @param dst The output matrix.
 */
__global__ void MatSquare(DeviceMatrix src, DeviceMatrix dst)
{
  // TODO: implement this kernel function
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
          i < src.n_rows;
          i += blockDim.x * gridDim.x) {
            for (int j = blockIdx.y * blockDim.y + threadIdx.y;
                    j < src.n_cols;
                    j += blockDim.y * gridDim.y) {
                      dst(i, j) = src(i, j) * src(i, j);
                    }
          }
}

/**
 * A CUDA kernel function that computes backpropagation for sigmoid function on
 * the GPU.
 *
 * @param da1 The upstream derivative matrix.
 * @param a1 The activation matrix.
 * @param dz1 The output derivative matrix.
 */
__global__ void MatSigmoidBackProp(DeviceMatrix da1, DeviceMatrix a1,
                                   DeviceMatrix dz1)
{
  /**
   * TODO: implement this kernel function
   * Hint: This kernel computes dz1 = da1 * a1 * (1 - a1), where * denotes
   * element-wise multiplication.
   */
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
          i < da1.n_rows;
          i += blockDim.x * gridDim.x) {
            for (int j = blockIdx.y * blockDim.y + threadIdx.y;
                    j < da1.n_cols;
                    j += blockDim.y * gridDim.y) {
                      dz1(i, j) = da1(i, j) * a1(i, j) * (1 - a1(i, j));
                    }
          }
}

__global__ void Warmup() {}

// +-*=+-*=+-*=+-*=+-*=+-*=+-*=+-*=+*-=+-*=+*-=+-*=+-*=+-*=+-*=+-*= //
//                       GPU kernel wrappers                        //
// +-*=+-*=+-*=+-*=+-*=+-*=+-*=+-*=+*-=+-*=+*-=+-*=+-*=+-*=+-*=+-*= //

void DSigmoid(DeviceMatrix src, DeviceMatrix dst)
{
  // TODO: implement this function
  dim3 block = {32, 32};
  dim3 grid = {(src.n_rows + block.x - 1) / block.x, (src.n_cols + block.y - 1) / block.y};
  MatSigmoid<<<grid, block>>>(src, dst);
  CHECK_LAUNCH("DSigmoid");
}

void DRepeatColVec(DeviceMatrix src, DeviceMatrix dst, int repeat)
{
  // TODO: implement this function
  dim3 block = {32, 32};
  dim3 grid = {(src.n_rows + block.x - 1) / block.x, (src.n_cols + block.y - 1) / block.y};
  MatRepeatColVec<<<grid, block>>>(src, dst, repeat);
  CHECK_LAUNCH("DRepeatColVec");
}

void DSum(DeviceMatrix src, DeviceMatrix dst, nn_real alpha, int axis)
{
  // TODO: implement this function
  int block = 32;
  int grid = 0;
  if (axis == 0)
  {
    grid = (src.n_cols + block - 1) / block;
  }
  else
  {
    grid = (src.n_rows + block - 1) / block;
  }
  MatSum<<<grid, block>>>(src, dst, alpha, axis);
  CHECK_LAUNCH("DSum");
}

void DSoftmax(DeviceMatrix src, DeviceMatrix dst, int axis)
{
  // TODO: implement this function
  int block = 32;
  int grid = 0;
  if (axis == 0)
  {
    grid = (src.n_cols + block - 1) / block;
  }
  else
  {
    grid = (src.n_rows + block - 1) / block;
  }
  MatSoftmax<<<grid, block>>>(src, dst, axis);

  CHECK_LAUNCH("DSoftmax");
}

void DCELoss(DeviceMatrix y_pred, DeviceMatrix y, DeviceMatrix loss)
{
  /**
   * TODO: implement this function
   * Hint: Initialize a temporary matrix T to store the loss and then call
   * MatCrossEntropyLoss. Call DSum twice to compute the sum of all elements
   * in T.
   */
  DeviceMatrix temp_loss1(y.n_rows, y.n_cols);
  DeviceMatrix temp_loss2(y.n_rows, 1);
  dim3 block = {32, 32};
  dim3 grid = {(y.n_rows + block.x - 1) / block.x, (y.n_cols + block.y - 1) / block.y};
  MatCrossEntropyLoss<<<grid, block>>>(y_pred, y, temp_loss1);
  DSum(temp_loss1, temp_loss2, 1, 1);
  DSum(temp_loss2, loss, 1, 0);
  CHECK_LAUNCH("DCELoss");
}

void DElemArith(DeviceMatrix A, DeviceMatrix B, nn_real alpha, nn_real beta)
{
  // TODO: implement this function
  dim3 block = {32, 32};
  dim3 grid = {(A.n_rows + block.x - 1) / block.x, (A.n_cols + block.y - 1) / block.y};
  MatElemArith<<<grid, block>>>(A, B, alpha, beta);

  CHECK_LAUNCH("DElemArith");
}

void DSquare(DeviceMatrix src, DeviceMatrix dst)
{
  // TODO: implement this function

  dim3 block = {32, 32};
  dim3 grid = {(src.n_rows + block.x - 1) / block.x, (src.n_cols + block.y - 1) / block.y};
  MatSquare<<<grid, block>>>(src, dst);

  CHECK_LAUNCH("DSquare");
}

void DSigmoidBackprop(DeviceMatrix da1, DeviceMatrix a1, DeviceMatrix dz1)
{
  // TODO: implement this function

  dim3 block = {32, 32};
  dim3 grid = {(da1.n_rows + block.x - 1) / block.x, (da1.n_cols + block.y - 1) / block.y};
  MatSigmoidBackProp<<<grid, block>>>(da1, a1, dz1);

  CHECK_LAUNCH("DSigmoidBackprop");
}

void DWarmup() { Warmup<<<1, 1>>>(); }
